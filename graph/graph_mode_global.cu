#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>  
#include <cstdint>
#include "error_handling.h"
#include "KernelForTest.cuh"

#define MEM_SIZE 1024 * 1024 * 8
#define VEC_SIZE 100

int main(){
    int threadsPerBlock = 1024;
    int Blocks = (MEM_SIZE / sizeof(int64_t) + threadsPerBlock - 1) / threadsPerBlock;

    hipStream_t stream1;
    RUNTIME_CHECK(hipStreamCreate(&stream1));

    int64_t* host_ptr;
    int64_t* device_ptr;
    RUNTIME_CHECK(hipHostAlloc(&host_ptr, MEM_SIZE, hipHostMallocDefault));
    RUNTIME_CHECK(hipMalloc(&device_ptr, MEM_SIZE));


    /* ----------------------------- 1. CUDA graph 捕获 --------------------------------- */
    hipGraph_t graph1;
    RUNTIME_CHECK(hipStreamBeginCapture(stream1, hipStreamCaptureModeGlobal));

    SingleVecKernel_int64_t<<< Blocks, threadsPerBlock, 0, stream1 >>>(device_ptr, VEC_SIZE, 100);
    SingleVecKernel_int64_t<<< Blocks, threadsPerBlock, 0, stream1 >>>(device_ptr + VEC_SIZE, VEC_SIZE, 200);
    SingleVecAddKernel_int64_t<<< Blocks, threadsPerBlock, 0, stream1 >>>(device_ptr, 2 * VEC_SIZE , 100);
   
    RUNTIME_CHECK(hipStreamEndCapture(stream1, &graph1));

    /* ----------------------------- 1. graph 结束捕获 --------------------------------- */

    /* ----------------------------- 2. CUDA graph 实例化并执行 ------------------------- */
    hipGraphExec_t graphExec1;
    RUNTIME_CHECK(hipGraphInstantiate(&graphExec1, graph1, NULL, NULL, 0));
    RUNTIME_CHECK(hipGraphLaunch(graphExec1, stream1));
    RUNTIME_CHECK(hipMemcpy(host_ptr, device_ptr, MEM_SIZE, hipMemcpyDeviceToHost));
    RUNTIME_CHECK(hipStreamSynchronize(stream1));
    
    /* ------------------------------------ 3. 验证结果 -------------------------------- */
    for(int i = 0; i < VEC_SIZE; i++){
        if (host_ptr[i] != 200){
            std::cout << host_ptr[i] << " ";
            exit(-1);
        }
    }
    std::cout << std::endl;
    for(int i = VEC_SIZE; i < 2 * VEC_SIZE; i++){
        if (host_ptr[i] != 300){
            std::cout << host_ptr[i] << " ";
            exit(-1);
        }
    }

    /* ----------------------------- 4. 释放资源 ----------------------------------- */
    RUNTIME_CHECK(hipHostFree(host_ptr));
    RUNTIME_CHECK(hipFree(device_ptr));
    RUNTIME_CHECK(hipStreamDestroy(stream1));
    RUNTIME_CHECK(hipGraphExecDestroy(graphExec1));
    RUNTIME_CHECK(hipGraphDestroy(graph1));

    return 0;

}