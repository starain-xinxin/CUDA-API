#include <iostream>
#include <hip/hip_runtime.h>

#define CUDA_CHECK_ERROR(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " << hipGetErrorString(err) << std::endl; \
        exit(1); \
    } \
}

__global__ void kernel(float* d_data) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < 1024) {
        d_data[idx] += 1.0f;
    }
}

void runCudaGraphRelaxed() {
    float* d_data;
    size_t size = 1024 * sizeof(float);

    // 创建 CUDA 流并设置非阻塞标志
    // ！！！！！！
    // 这里不能设置为阻塞模式，否则这将导致捕获出错
    // ！！！！！！
    hipStream_t stream;
    CUDA_CHECK_ERROR(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    // 开始捕获流
    CUDA_CHECK_ERROR(hipStreamBeginCapture(stream, hipStreamCaptureModeRelaxed));

    // 在捕获图中分配内存
    CUDA_CHECK_ERROR(hipMalloc(&d_data, size));

    // 初始化数据
    CUDA_CHECK_ERROR(hipMemset(d_data, 0, size));

    // 捕获内核调用
    kernel<<<4, 256, 0, stream>>>(d_data);

    // 捕获结束
    hipGraph_t graph;
    CUDA_CHECK_ERROR(hipStreamEndCapture(stream, &graph));

    // 实例化图
    hipGraphExec_t graphExec;
    CUDA_CHECK_ERROR(hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0));

    // 重放图
    CUDA_CHECK_ERROR(hipGraphLaunch(graphExec, stream));
    CUDA_CHECK_ERROR(hipStreamSynchronize(stream));

    // 清理资源
    CUDA_CHECK_ERROR(hipFree(d_data));
    CUDA_CHECK_ERROR(hipGraphExecDestroy(graphExec));
    CUDA_CHECK_ERROR(hipGraphDestroy(graph));
    CUDA_CHECK_ERROR(hipStreamDestroy(stream));
}


int main() {
    std::cout << "Running with Relaxed mode..." << std::endl;
    runCudaGraphRelaxed();
    std::cout << "CUDA graph with Relaxed mode executed successfully." << std::endl;
    return 0;
}
