#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include <cstdint>
#include "error_handling.h"
#include "KernelForTest.cuh"

#define HOST_MEM_SIZE 1024 * 1024 * 8
#define VEC_SIZE 100

int main(){
    /* 0. 线程数，显存等预处理 */
    int threadsPerBlock = 1024;
    int blocks = (HOST_MEM_SIZE / sizeof(int64_t) + threadsPerBlock - 1) / threadsPerBlock;

    int64_t* host_ptr;
    RUNTIME_CHECK(hipHostAlloc(&host_ptr, HOST_MEM_SIZE, hipHostMallocDefault));
    int64_t* device_ptr;
    RUNTIME_CHECK(hipMalloc(&device_ptr, HOST_MEM_SIZE));  

    /* ------------------------------------ 1. 创建图 ------------------------------------ */
    hipGraph_t graph1;
    // hipGraphNode_t MallocNode, Kernel_B, C, D, E;
    RUNTIME_CHECK(hipGraphCreate(&graph1, 0));

    hipGraphNode_t Kernel_A;
    hipKernelNodeParams Param_A;
    Param_A.blockDim = dim3(threadsPerBlock);
    Param_A.gridDim = dim3(blocks);
    Param_A.func = (void*)SingleVecKernel_int64_t;
    Param_A.sharedMemBytes = 0; 
    Param_A.extra = NULL; 
    int64_t* A_p_vec = device_ptr; int A_N = VEC_SIZE; int64_t A_number = 100;
    void* KernelParams_A[3] = {&A_p_vec, &A_N, &A_number};
    Param_A.kernelParams = KernelParams_A;
    RUNTIME_CHECK(hipGraphAddKernelNode(&Kernel_A, graph1, NULL, 0, &Param_A));

    hipGraphNode_t Kernel_B;
    hipKernelNodeParams Param_B;
    Param_B.blockDim = dim3(threadsPerBlock);
    Param_B.gridDim = dim3(blocks);
    Param_B.func = (void*)SingleVecKernel_int64_t;
    Param_B.sharedMemBytes = 0; 
    Param_B.extra = NULL; 
    int64_t* B_p_vec = device_ptr + VEC_SIZE; int B_N = VEC_SIZE; int64_t B_number = 200;
    void* KernelParams_B[3] = {&B_p_vec, &B_N, &B_number};
    Param_B.kernelParams = KernelParams_B;
    RUNTIME_CHECK(hipGraphAddKernelNode(&Kernel_B, graph1, NULL, 0, &Param_B));

    hipGraphNode_t Kernel_C;
    hipKernelNodeParams Param_C;
    Param_C.blockDim = dim3(threadsPerBlock);
    Param_C.gridDim = dim3(blocks);
    Param_C.func = (void*)SingleVecAddKernel_int64_t;
    Param_C.sharedMemBytes = 0; 
    Param_C.extra = NULL; 
    int64_t* C_p_vec = device_ptr; int C_N = 2 * VEC_SIZE; int64_t C_number = 100;
    void* KernelParams_C[3] = {&C_p_vec, &C_N, &C_number};
    Param_C.kernelParams = KernelParams_C;
    RUNTIME_CHECK(hipGraphAddKernelNode(&Kernel_C, graph1, NULL, 0, &Param_C));

    hipGraphNode_t MemCpy_D;
    RUNTIME_CHECK(hipGraphAddMemcpyNode1D(&MemCpy_D, graph1, NULL, 0, host_ptr, device_ptr, HOST_MEM_SIZE, hipMemcpyDeviceToHost));

    // 添加依赖
    RUNTIME_CHECK(hipGraphAddDependencies(graph1, &Kernel_A, &Kernel_C, 1)); // A -> C
    RUNTIME_CHECK(hipGraphAddDependencies(graph1, &Kernel_B, &Kernel_C, 1)); // B -> C
    RUNTIME_CHECK(hipGraphAddDependencies(graph1, &Kernel_C, &MemCpy_D, 1)); // c -> D



     /* ------------------------------------ 2. 实例化图 ------------------------------------ */
    hipGraphExec_t graphExec1;
    RUNTIME_CHECK(hipGraphInstantiate(&graphExec1, graph1, NULL, NULL, 0));
    hipStream_t stream1;
    RUNTIME_CHECK(hipStreamCreate(&stream1));

    /* ------------------------------------ 3. 执行图 ------------------------------------- */
    hipGraphLaunch(graphExec1, stream1);
    hipStreamSynchronize(stream1);

    /* ------------------------------------ 4. 验证结果 ---------- */
    for(int i = 0; i < VEC_SIZE; i++){
        assert(host_ptr[i] == 200 && "Error: The result is not 200");
    }
    for(int i = VEC_SIZE; i < 2 * VEC_SIZE; i++){
        assert(host_ptr[i] == 300 && "Error: The result is not 300");
    }

    /* ------------------------------------ 5. 释放资源 ------------------------------------ */
    RUNTIME_CHECK(hipGraphExecDestroy(graphExec1));
    RUNTIME_CHECK(hipStreamDestroy(stream1));
    RUNTIME_CHECK(hipGraphDestroy(graph1));
    RUNTIME_CHECK(hipFree(device_ptr));
    RUNTIME_CHECK(hipHostFree(host_ptr));


    return 0;
}
