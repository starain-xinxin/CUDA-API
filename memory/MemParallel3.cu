#include "hip/hip_runtime.h"
// 这个文件证明了并发的调用vmm-api并不能真正提高性能。这里的瓶颈在于GPU的调度能力，而非cpu。

#include <cstddef>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "error_handling.h"
#include "KernelForTest.cuh"
#include <chrono>
#include <memory>
#include <vector>
#include <cstddef>
#include <mutex>
#include <fstream>
#include <ctime>
#include <iomanip>
#include <sstream>
#include <string>
#include <thread>

#define NUM_KERNELS 30000000

// 定义一个全局互斥锁以确保线程安全（如果您的应用程序是多线程的）
std::mutex log_mutex;

/**
 * @brief 记录当前时间，并根据参数决定输出到控制台或文件
 * 
 * @param label 描述当前时间点的字符串
 * @param output 输出位置标识符：0 表示输出到控制台，1 表示输出到文件
 */
void log_time(const std::string& label, int output) {
    // 获取当前时间点
    auto now = std::chrono::system_clock::now();

    // 转换为 time_t 以便格式化
    std::time_t now_time_t = std::chrono::system_clock::to_time_t(now);

    // 转换为本地时间结构
    std::tm* local_tm = std::localtime(&now_time_t);

    // 获取毫秒部分
    auto now_ms = std::chrono::time_point_cast<std::chrono::milliseconds>(now);
    auto value = now_ms.time_since_epoch();
    long duration = value.count() % 1000;

    // 格式化时间字符串
    std::ostringstream oss;
    oss << std::put_time(local_tm, "%Y-%m-%d %H:%M:%S") 
        << "." << std::setfill('0') << std::setw(3) << duration;

    std::string formatted_time = oss.str();

    // 构造日志消息
    std::ostringstream log_message;
    log_message << "[" << formatted_time << "] " << label;

    // 使用互斥锁确保线程安全
    std::lock_guard<std::mutex> guard(log_mutex);

    if (output == 0) {
        // 仅打印到控制台
        std::cout << log_message.str() << std::endl;
    }
    else if (output == 1) {
        // 仅打印到日志文件
        std::ofstream log_file("log.txt", std::ios_base::app); // 以追加模式打开
        if (log_file.is_open()) {
            log_file << log_message.str() << std::endl;
            log_file.close();
        } else {
            std::cerr << "无法打开 log.txt 文件进行写入。" << std::endl;
        }
    }
    else {
        // 无效的输出标识符
        std::cerr << "Invalid output identifier: " << output << ". Use 0 for console or 1 for file." << std::endl;
    }
}

hipError_t setMemAccess(void* ptr, std::uint64_t size, int current_device_in = -1)
{
    int current_device = 0;
    // if(current_device == -1) {
        // DRIVE_CHECK(hipCtxGetDevice(&current_device));
    // }

    hipMemAccessDesc accessDesc = {};
    accessDesc.location.type = hipMemLocationTypeDevice;
    accessDesc.location.id = current_device;
    accessDesc.flags = hipMemAccessFlagsProtReadWrite;
    hipError_t result = hipMemSetAccess((hipDeviceptr_t)ptr, size, &accessDesc, 1); 
    return result;
}

struct phy_block {
    hipDeviceptr_t ptr;
    hipMemGenericAllocationHandle_t alloc_handle;
    phy_block(hipDeviceptr_t ptr_, hipMemGenericAllocationHandle_t alloc_handle_) : 
        ptr(ptr_), alloc_handle(alloc_handle_) {}
    ~phy_block(){
        DRIVE_CHECK(hipMemRelease(alloc_handle)); 
    }
};

struct Block{
    hipDeviceptr_t ptr;
    size_t size;
    std::vector<std::shared_ptr<phy_block>> phy_blocks;

    Block(hipDeviceptr_t ptr_, size_t size_, std::vector<std::shared_ptr<phy_block>>&& phy_blocks_):
        ptr(ptr_), size(size_), phy_blocks(std::move(phy_blocks_)) {}

    ~Block(){
        // 解除映射
        DRIVE_CHECK(hipMemUnmap((hipDeviceptr_t)ptr, size));
        // 释放虚拟地址
        DRIVE_CHECK(hipMemAddressFree(hipDeviceptr_t(ptr), size));
    }
    
};

/* 获取内存分配所需的最小分配粒度 */
size_t getGranularitySize()
{
    static size_t granularity_ = 0;

    if(granularity_ == 0) {
        // int current_device;
        // DRIVE_CHECK(hipCtxGetDevice(&current_device));  // 获取当前上下文的设备ID

        hipMemAllocationProp prop = {};
        prop.type = hipMemAllocationTypePinned;  // 固定内存
        prop.location.type = hipMemLocationTypeDevice;
        prop.location.id = 0;  // 设备ID

        DRIVE_CHECK(hipMemGetAllocationGranularity(&granularity_, &prop, hipMemAllocationGranularityMinimum));
    }

    return granularity_;
}

std::shared_ptr<Block> getBlock(size_t size, size_t granularity){
    int num_phy = size / granularity;
    // 1. reserve 一段虚拟内存
    hipDeviceptr_t ptr;
    DRIVE_CHECK(hipMemAddressReserve(&ptr, size, 0, 0, 0));
    // 2. 创建物理内存，并将其映射到虚拟内存
    std::vector<std::shared_ptr<phy_block>> tmp_phy_blocks;
    for(int i = 0; i < num_phy; i++){
        hipMemGenericAllocationHandle_t alloc_handle;      // 固定内存块句柄
        hipMemAllocationProp prop = {};
        prop.type = hipMemAllocationTypePinned;      
        prop.location.type = hipMemLocationTypeDevice;   
        prop.location.id = 0;  // 设备ID
        DRIVE_CHECK(hipMemCreate(&alloc_handle, granularity, &prop, 0));   // 创建固定内存块
        std::shared_ptr<phy_block> phy = std::make_shared<phy_block> (ptr + i * granularity, alloc_handle);
        tmp_phy_blocks.push_back(std::move(phy));
        // 3. 映射到虚拟内存
        auto  block_ptr = (void*) ( ((char*)ptr) + (i * granularity));
        hipDeviceptr_t device_ptr = (hipDeviceptr_t)block_ptr;
        DRIVE_CHECK(hipMemMap(device_ptr, granularity,0ULL, alloc_handle, 0ULL));
        // 4. 设置权限
        hipError_t err = setMemAccess((char*)(ptr) + i * granularity, granularity);
    }
    std::shared_ptr<Block> new_block = std::make_shared<Block>(ptr, size, std::move(tmp_phy_blocks));
    return new_block;
}

void mem_parallel1(){
    // 测试单线程的vmm-api申请与释放内存的代价
    auto t0 = std::chrono::steady_clock::now();
    log_time("Start testing vmm-api calls 1", 0);
    // 显存块大小：5GB，10GB，8GB，16GB，2GB，6GB
    std::vector<std::uint64_t> MemBlockSizes = {
    5ULL * 1024 * 1024 * 1024,
    10ULL * 1024 * 1024 * 1024,
    8ULL * 1024 * 1024 * 1024,
    };
    // 获取显存分配粒度
    size_t granularity = 0;
    granularity = getGranularitySize();
    // 存放Block的池
    std::vector<std::shared_ptr<Block>> pool;
    
    for (auto& MemBlockSize : MemBlockSizes){
        std::shared_ptr<Block> new_block = getBlock(MemBlockSize, granularity);
        pool.push_back(new_block);
        // std::cout << "Allocating " << MemBlockSize << " bytes of memory, " << new_block->phy_blocks.size() << " physical blocks" << std::endl;
    }
    // 释放所有Block
    for (auto& block : pool){
        block.reset();
    }
    auto t1 = std::chrono::steady_clock::now();
    log_time("End testing vmm-api calls 1", 0);
    using Ms = std::chrono::duration<double, std::milli>;
    Ms vmm_api_time = t1 - t0;
    std::cout << "Total time for vmm-api memory allocation and deallocation 1: " << vmm_api_time.count() << " ms" << std::endl;
}

void mem_parallel2(){
    // 测试单线程的vmm-api申请与释放内存的代价
    auto t0 = std::chrono::steady_clock::now();
    log_time("Start testing vmm-api calls 2", 0);
    // 显存块大小：5GB，10GB，8GB，16GB，2GB，6GB
    std::vector<std::uint64_t> MemBlockSizes = {
    16ULL * 1024 * 1024 * 1024,
    2ULL * 1024 * 1024 * 1024,
    6ULL * 1024 * 1024 * 1024
    };
    // 获取显存分配粒度
    size_t granularity = 0;
    granularity = getGranularitySize();
    // 存放Block的池
    std::vector<std::shared_ptr<Block>> pool;
    
    for (auto& MemBlockSize : MemBlockSizes){
        std::shared_ptr<Block> new_block = getBlock(MemBlockSize, granularity);
        pool.push_back(new_block);
        // std::cout << "Allocating " << MemBlockSize << " bytes of memory, " << new_block->phy_blocks.size() << " physical blocks" << std::endl;
    }
    // 释放所有Block
    for (auto& block : pool){
        block.reset();
    }
    auto t1 = std::chrono::steady_clock::now();
    log_time("End testing vmm-api calls 2", 0);
    using Ms = std::chrono::duration<double, std::milli>;
    Ms vmm_api_time = t1 - t0;
    std::cout << "Total time for vmm-api memory allocation and deallocation 2: " << vmm_api_time.count() << " ms" << std::endl;
}

int main(){
    // setup
    int N = 1024;
    float *d_A, *d_B, *d_C;
    // 分配设备端内存
    RUNTIME_CHECK(hipMalloc(&d_A, N * N * sizeof(float)));
    RUNTIME_CHECK(hipMalloc(&d_B, N * N * sizeof(float)));
    RUNTIME_CHECK(hipMalloc(&d_C, N * N * sizeof(float)));

    // 定义 CUDA 线程块和网格大小
    dim3 blockDim(16, 16);  // 每个线程块包含 16x16 个线程
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);

    // 测试单线程计算NUN_KERNELS次核函数的代价
    using Ms = std::chrono::duration<double, std::milli>;
    Ms sigle_thread_time_for_100kernel_ = Ms(0);
    auto t0 = std::chrono::steady_clock::now();
    auto tt_pre = std::chrono::steady_clock::now();
    
    std::thread thread1(mem_parallel1);
    std::thread thread2(mem_parallel2);
    

    log_time("Start testing kernel calls", 0);
    for (int i = 0; i < NUM_KERNELS; i++){
        matrixMultiply<<<blockDim, gridDim>>>(d_A, d_B, d_C, N);
        // if (i%100 == 0 && i!= 0){
        //     hipDeviceSynchronize();
        //     auto tt = std::chrono::steady_clock::now();
        //     std::cout << "Kernel " << i << " finished in " << std::chrono::duration_cast<Ms>(tt - tt_pre).count() << " ms" << std::endl;
        //     tt_pre = tt;
        // }
    }
    hipDeviceSynchronize();
    log_time("End testing kernel calls", 0);

    auto t1 = std::chrono::steady_clock::now();
    sigle_thread_time_for_100kernel_ = t1 - t0;
    std::cout << "Total time for " << NUM_KERNELS << " kernel calls: " << sigle_thread_time_for_100kernel_.count() << " ms" << std::endl;
    
    thread1.join();
    thread2.join();
    
}
