#include <iostream>
#include <hip/hip_runtime.h>
#include "error_handling.h"
#include "KernelForTest.cuh"
#include <cstdint>

// 对于某个流，异步上传计算任务
void upload_task(hipStream_t stream, int64_t* p_device_memory, size_t num_data){
    // 计算需要的线程数
    int num_threads = 128;
    int num_blocks = (num_data + num_threads - 1) / num_threads;
    RUNTIME_CHECK(hipMalloc(&p_device_memory, num_data * sizeof(int64_t)));
    SingleVecKernel_int64_t <<<num_blocks, num_threads, 0, stream>>> (p_device_memory, num_data, 666);
}

int main() {

    // 获取优先级范围
    int leastPriority = 0;
    int greatestPriority = 0;
    RUNTIME_CHECK(hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority));
    std::cout << "Least priority: " << leastPriority << std::endl;
    std::cout << "Greatest priority: " << greatestPriority << std::endl;

    // 创建四个流,两个最高，两个最低
    hipStream_t streams[4];
    RUNTIME_CHECK(hipStreamCreateWithPriority(&streams[0], hipStreamNonBlocking, greatestPriority));
    RUNTIME_CHECK(hipStreamCreateWithPriority(&streams[3], hipStreamNonBlocking, greatestPriority));
    RUNTIME_CHECK(hipStreamCreateWithPriority(&streams[1], hipStreamNonBlocking, leastPriority));
    RUNTIME_CHECK(hipStreamCreateWithPriority(&streams[2], hipStreamNonBlocking, leastPriority));
    std::cout << "优先级排序：0 = 3 > 1 = 2" << std::endl; 

    // 准备指针与任务配置
    int64_t* p_list[4];
    size_t num_data = 128 * 1024 * 1024;

    // 上传任务
    for (int i = 0; i < 4; i++) {
        upload_task(streams[i], p_list[i], num_data);
    }

    // 非阻塞的询问流是否完成
    bool is_done[4] = {false, false, false, false};
    int priority_sort[4];
    int no_stream = 0;
    hipError_t error[4];
    while(no_stream < 4){    // 轮询流是否完成
        for (int i = 0; i < 4; i++) {
            if (!is_done[i]){
                error[i] = hipStreamQuery(streams[i]);
                if (error[i] == hipSuccess) {
                    std::cout << "Stream " << i << " is done." << std::endl;
                    is_done[i] = true;
                    priority_sort[no_stream] = i;
                    no_stream++;
                }
            }
        }
    }

    // 打印实测优先级排序
    std::cout << "实测优先级排序：" << priority_sort[0] << " " << priority_sort[1] << " " << priority_sort[2] << " " << priority_sort[3] << " "<< std::endl;

    // 释放资源
    for (int i = 0; i < 4; i++) {
        RUNTIME_CHECK(hipStreamDestroy(streams[i]));
        RUNTIME_CHECK(hipFree(p_list[i]));
    }
    
    return 0;
}