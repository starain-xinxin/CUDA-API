#include <iostream>
#include "KernelForTest.cuh"
#include "error_handling.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <chrono>
#include <cstdint>

#define NUM_STREAMS 8
#define HOST_MEM_SIZE 1024 * 1024 * 1024 // 1GB

int main() {
    // 创建多个流
    std::vector<hipStream_t> stream_list(NUM_STREAMS);
    for(int i = 0; i < NUM_STREAMS; i++) {
        RUNTIME_CHECK(hipStreamCreateWithFlags(&stream_list[i], hipStreamNonBlocking));
    }

    // 创建一块主机的页锁定内存，并把所有的数据写为 666
    constexpr size_t DATA_SIZE = sizeof(int64_t);
    int64_t* p_host_mem;
    RUNTIME_CHECK(hipHostMalloc(&p_host_mem, HOST_MEM_SIZE));
    for(size_t i = 0; i < HOST_MEM_SIZE / DATA_SIZE; i++) {
        p_host_mem[i] = 666;
    }
    // 测试写入成功与否
    for(size_t i = 0; i < HOST_MEM_SIZE / DATA_SIZE; i++) {
        if(p_host_mem[i] != 666) {
            std::cout << "主机内存写入失败" << std::endl;
            return 0;
        }
    }

    // 创建一块设备的页锁定内存，并把所有的数据写为 666
    int64_t* p_device_mem;
    RUNTIME_CHECK(hipMalloc(&p_device_mem, HOST_MEM_SIZE));

    // ------------------------------------------------------ 1. 单流执行 ------------------------------------------------------ //
    std::chrono::high_resolution_clock::time_point start_time, end_time;
    start_time = std::chrono::high_resolution_clock::now();
    // 拷贝，执行，拷贝
    RUNTIME_CHECK(hipMemcpyAsync(p_device_mem, p_host_mem, HOST_MEM_SIZE, 
                                  hipMemcpyHostToDevice, stream_list[0]));
    int threadsPerBlock = 1024;
    int blocks = (HOST_MEM_SIZE / sizeof(int64_t) + threadsPerBlock - 1) / threadsPerBlock;
    SingleVecKernel_int64_t<<<blocks, threadsPerBlock, 0, stream_list[0]>>>(p_device_mem, HOST_MEM_SIZE / sizeof(int64_t), 777);
    RUNTIME_CHECK(hipMemcpyAsync(p_host_mem, p_device_mem, HOST_MEM_SIZE,
                                  hipMemcpyDeviceToHost, stream_list[0]));
    // 同步计时
    RUNTIME_CHECK(hipStreamSynchronize(stream_list[0]));
    end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsed_time = end_time - start_time;
    // 核函数报错处理
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
    }
    // 验证结果
    bool single_flag = true;
    for(size_t i = 0; i < HOST_MEM_SIZE / DATA_SIZE; i++) {
        if(p_host_mem[i] != 777) 
            single_flag = false;
    }
    if (single_flag)
        std::cout << "单流执行用时: " << elapsed_time.count() << " ms" << std::endl;
    else
        std::cout << "单流执行结果验证失败" << std::endl;
    // ------------------------------------------------------ 1. 单流执行 ------------------------------------------------------ //


    // ------------------------------------------------------ 2. 多流执行 ------------------------------------------------------ //
    start_time = std::chrono::high_resolution_clock::now();
    size_t chunk_size = HOST_MEM_SIZE / NUM_STREAMS; // 每个流处理的数据块大小
    // 拷贝，执行，拷贝
    for(int i = 0; i < NUM_STREAMS; i++) {
        // 计算当前流需要传递的指针
        int64_t* p_host_chunk = p_host_mem + i * chunk_size / DATA_SIZE;  
        int64_t* p_device_chunk = p_device_mem + i * chunk_size / DATA_SIZE;  

        // 异步将数据从主机拷贝到设备
        RUNTIME_CHECK(hipMemcpyAsync(p_device_chunk, p_host_chunk, chunk_size, 
                                    hipMemcpyHostToDevice, stream_list[i]));
        
        // 动态计算块数和每块线程数
        int threadsPerBlock = 1024;  // 你可以根据实际情况调整
        int blocks = (chunk_size / sizeof(int64_t) + threadsPerBlock - 1) / threadsPerBlock;

        // 核函数调用，指定每个流的网格和块大小
        SingleVecKernel_int64_t<<<blocks, threadsPerBlock, 0, stream_list[i]>>>(p_device_chunk, chunk_size / DATA_SIZE, 888);

        // 异步将数据从设备拷贝到主机
        RUNTIME_CHECK(hipMemcpyAsync(p_host_chunk, p_device_chunk, chunk_size,
                                    hipMemcpyDeviceToHost, stream_list[i]));
    }
    // 同步所有流并计时
    for(int i = 0; i < NUM_STREAMS; i++) {
        RUNTIME_CHECK(hipStreamSynchronize(stream_list[i]));
    }
    end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsed_time2 = end_time - start_time;
    // 核函数报错处理
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
    }
    // 验证结果
    bool multi_flag = true;
    for(size_t i = 0; i < HOST_MEM_SIZE / DATA_SIZE; i++) {
        if(p_host_mem[i] != 888) 
            multi_flag = false;
    }
    if (multi_flag)
        std::cout << "多流执行用时: " << elapsed_time2.count() << " ms" << std::endl;
    else{
        std::cout << "多流执行结果验证失败" << std::endl;
        for(size_t i = HOST_MEM_SIZE / DATA_SIZE - 1; i > HOST_MEM_SIZE / DATA_SIZE - 21; i--) {
            std::cout << p_host_mem[i] << " ";
        }
        std::cout << std::endl;
    }
    // ------------------------------------------------------ 2. 多流执行 ------------------------------------------------------ //

    // 释放资源
    RUNTIME_CHECK(hipHostFree(p_host_mem));
    RUNTIME_CHECK(hipFree(p_device_mem));
    for(int i = 0; i < NUM_STREAMS; i++) {
        RUNTIME_CHECK(hipStreamDestroy(stream_list[i]));
    }

    // 加速比计算
    std::cout << "加速比: " << elapsed_time.count() / elapsed_time2.count() << " 倍" << std::endl;


    return 0;
}
