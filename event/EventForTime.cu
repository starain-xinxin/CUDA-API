#include <iostream>
#include "KernelForTest.cuh"
#include "error_handling.h"
#include <hip/hip_runtime.h>


int main() {
    hipStream_t stream_id = 0;
    hipEvent_t start, stop;
    RUNTIME_CHECK(hipEventCreate(&start));
    RUNTIME_CHECK(hipEventCreate(&stop));
   
    

    int N = 4;  
    float A[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16};  
    float B[] = {16, 15, 14, 13, 12, 11, 10, 9, 8, 7, 6, 5, 4, 3, 2, 1}; 
    float C[16]; 
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * N * sizeof(float));
    hipMalloc(&d_B, N * N * sizeof(float));
    hipMalloc(&d_C, N * N * sizeof(float));
    hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);


    RUNTIME_CHECK(hipEventRecord(start, stream_id));
    // ...
    // 启动 CUDA 核函数
    matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    // ...

    RUNTIME_CHECK(hipEventRecord(stop, stream_id));
    RUNTIME_CHECK(hipEventSynchronize(stop));
    float milliseconds = 0;
    RUNTIME_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "Kernel time: " << milliseconds << " ms" << std::endl;



    hipDeviceSynchronize();
    hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    


    return 0;
}
