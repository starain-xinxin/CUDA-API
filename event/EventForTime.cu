#include <iostream>
#include "KernelForTest.cuh"
#include "error_handling.h"
#include <hip/hip_runtime.h>

#define NUM_CALL_KERNEL 100000

int main() {
    hipStream_t stream_id;
    hipStreamCreate(&stream_id);
        
    hipEvent_t start, stop;
    RUNTIME_CHECK(hipEventCreate(&start));
    RUNTIME_CHECK(hipEventCreate(&stop));
    

    int N = 4;  
    float A[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16};  
    float B[] = {16, 15, 14, 13, 12, 11, 10, 9, 8, 7, 6, 5, 4, 3, 2, 1}; 
    float C[16]; 
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * N * sizeof(float));
    hipMalloc(&d_B, N * N * sizeof(float));
    hipMalloc(&d_C, N * N * sizeof(float));
    hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);


    RUNTIME_CHECK(hipEventRecord(start, stream_id));       // 利用start事件打上标记
    // ...
    // 启动 CUDA 核函数
    for (int i = 0; i < NUM_CALL_KERNEL; i++) {    
        matrixMultiply<<<blocksPerGrid, threadsPerBlock, 0, stream_id>>>(d_A, d_B, d_C, N);
    }
    // ...

    RUNTIME_CHECK(hipEventRecord(stop, stream_id));       // 利用stop事件打上标记
    RUNTIME_CHECK(hipEventSynchronize(stop));             // 阻塞式同步stop事件完成

    // 计算Kernel运行时间
    float milliseconds = 0;
    RUNTIME_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << NUM_CALL_KERNEL <<"次 kernel 用时: " << milliseconds << " ms" << std::endl;
    
    // 传输结果，释放资源等
    hipDeviceSynchronize();
    hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    return 0;
}
